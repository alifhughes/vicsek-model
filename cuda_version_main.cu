#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <SDL.h>
#include ""
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>

#define NUM_PARTICLES 1000
#define SHARED_BUFFER_SIZE 1000
#define SCREEN_W 600
#define SCREEN_H 600
#define LINE_LEN 5
#define SPEED 0.25f
#define RADIUS 0.25f
#define PHASE_LAG 1.53f
#define COUPLING 1.0f
#define DT 0.1f

//#define USE_SHARED_BUFFER

__device__ float RADIUS_SQ = RADIUS*RADIUS;

static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


typedef struct {
	float x, y, phi, dphi;

} particle_t;

__global__ void timestepKernel(particle_t* particles)
{


#ifdef USE_SHARED_BUFFER
	__shared__ particle_t sharedParticles[SHARED_BUFFER_SIZE];
#endif

	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < NUM_PARTICLES)
	{
		// Get particle to be updated by this thread.
		particle_t particle = particles[id];
		float dphi = 0.0f;
		float nearCount = 0.0f;

#ifdef USE_SHARED_BUFFER
		////
		// Shared memory version.
		////
		for (int i = 0; i < NUM_PARTICLES / SHARED_BUFFER_SIZE; i++) {
			// In each phase (i), one thread loads one particle into shared memory.
			sharedParticles[threadIdx.x] = particles[i*SHARED_BUFFER_SIZE + threadIdx.x];

		    //__syncthreads(); // Dangerous - make sure block size evenly divides particle count.

			// Find nearby particles and update direction.
			for (int j = 0; j < SHARED_BUFFER_SIZE; j++) {
				particle_t other = sharedParticles[j];

				float dx = particle.x - other.x;
				float dy = particle.y - other.y;
				if (dx*dx + dy*dy < RADIUS_SQ)
				{
					dphi += sin(other.phi - particle.phi - PHASE_LAG);
					nearCount += 1.0f;
				}
			}
		}
#else
		////
		// Non-shared memory version.
		////
		for (int j = 0; j < NUM_PARTICLES; j++) {
			particle_t other = particles[j];

			float dx = particle.x - other.x;
			float dy = particle.y - other.y;
			if (dx*dx + dy*dy < RADIUS_SQ)
			{
				dphi += sin(other.phi - particle.phi - PHASE_LAG);
				nearCount += 1.0f;
			}
		}
#endif
		particle.x += SPEED*DT*cos(particle.phi);
		particle.y += SPEED*DT*sin(particle.phi);

		// Wrap around at edges.
		if (particle.x > 1) particle.x -= 1;
		if (particle.x < 0) particle.x += 1;
		if (particle.y > 1) particle.y -= 1;
		if (particle.y < 0) particle.y += 1;

		// Update direction if there were any interactions.
		if (nearCount > 0) {
			particle.phi += DT*(COUPLING / nearCount)*dphi;
		}

		// Write new particle properties back to global memory.
		particles[id] = particle;
	}
}

int main(int argc, char** argv)
{
	// Allocate host buffer to store particle states.
	particle_t* particles;
	hipHostMalloc(&particles, sizeof(particle_t)*NUM_PARTICLES);

	// Random initial positions / directions.
	for (int i = 0; i < NUM_PARTICLES; i++)
	{
		particles[i].x = (float)rand() / (float)RAND_MAX;
		particles[i].y = (float)rand() / (float)RAND_MAX;
		particles[i].phi = ((float)rand() / (float)RAND_MAX) * 2 * M_PI;
	}

	// Initialize SDL and create window.
	if (SDL_Init(SDL_INIT_EVERYTHING) < 0) {
		perror("Error initializing SDL.");
		exit(1);
	}
	SDL_Window* window = SDL_CreateWindow("Vicsek",
		SDL_WINDOWPOS_UNDEFINED,
		SDL_WINDOWPOS_UNDEFINED,
		SCREEN_W, SCREEN_H,
		SDL_WINDOW_OPENGL);
	SDL_Renderer* renderer = SDL_CreateRenderer(window, -1, 0);

	// Allocate device memory for particles.
	particle_t* dev_particles;
	HANDLE_ERROR(hipMalloc((void**)&dev_particles,
		NUM_PARTICLES * sizeof(particle_t)));

	HANDLE_ERROR(hipMemcpy(dev_particles,
		particles,
		NUM_PARTICLES*sizeof(particle_t),
		hipMemcpyHostToDevice));

	int threadsPerBlock = SHARED_BUFFER_SIZE;
	dim3 gridSize(NUM_PARTICLES / threadsPerBlock);
	dim3 blockSize(threadsPerBlock);

	SDL_Event event;
	int running = 1;
	unsigned int lastTicks = SDL_GetTicks();
	float t = 0;

	int prePause = 2;
	float totalUpdate = 0;
	float totalRender = 0;
	int counter = 0;

	while (running)
	{

		unsigned int ticks = SDL_GetTicks();
		float dt = (ticks - lastTicks) / 1000.0f;
		lastTicks = ticks;

		while (SDL_PollEvent(&event))
		{
			if (event.type == SDL_QUIT)
			{
				running = 0;
				break;
			}

		}

		// Update particles using the GPU.
		timestepKernel << <gridSize, blockSize >> >(dev_particles);

		// Copy particle positions back to host.
		HANDLE_ERROR(hipMemcpy(particles,
			dev_particles,
			NUM_PARTICLES*sizeof(particle_t),
			hipMemcpyDeviceToHost));
		t += DT;

		unsigned int endUpdateTicks = SDL_GetTicks();

		// Draw particles.
		SDL_SetRenderDrawColor(renderer, 0, 0, 0, SDL_ALPHA_OPAQUE);
		SDL_RenderClear(renderer);
		for (int i = 0; i < NUM_PARTICLES; i++)
		{

			SDL_SetRenderDrawColor(renderer, 255, 255, 255, SDL_ALPHA_OPAQUE);
			SDL_RenderDrawLine(renderer,
				particles[i].x*SCREEN_W,
				particles[i].y*SCREEN_H,
				particles[i].x*SCREEN_W + LINE_LEN*cos(particles[i].phi),
				particles[i].y*SCREEN_H + LINE_LEN*sin(particles[i].phi));
		}
		SDL_RenderPresent(renderer);
		unsigned int endRenderTicks = SDL_GetTicks();


		std::cout << t << ": Update took " << endUpdateTicks - ticks << "ms. Draw took " << endRenderTicks - endUpdateTicks << "ms." << std::endl;
		 
		// Update the total amount of time
		totalUpdate += endUpdateTicks - ticks;
 		totalRender += endRenderTicks - endUpdateTicks;
		counter++;

		// Stop after 50 updates NOT WORKING YET AS CAN'T COMPARE FLOATS
		if (counter == 500) {

			break;
		}

	}

	// Get the average update and render time across the 50 updates
	float averageUpdate = totalUpdate / 500;
	float averageRender = totalRender / 500;

	std::cout << averageUpdate << " " << averageRender << std::endl;

	return 0;
}
